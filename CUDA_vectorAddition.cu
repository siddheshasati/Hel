#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(float *A, float *B, float *C, int N) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    int N = 1 << 20; // 1 million elements
    size_t size = N * sizeof(float);

    // Host vectors
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize vectors
    for (int i = 0; i < N; i++) {
        h_A[i] = i * 1.0f;
        h_B[i] = i * 2.0f;
    }

    // Device vectors
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy data to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print first 5 results
    for (int i = 0; i < 5; i++)
        std::cout << h_A[i] << " + " << h_B[i] << " = " << h_C[i] << std::endl;

    // Cleanup
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);

    return 0;
}
