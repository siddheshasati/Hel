#include <iostream>
#include <hip/hip_runtime.h>
#define N 16

__global__ void matMulKernel(float *A, float *B, float *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0;
    if (row < n && col < n) {
        for (int k = 0; k < n; ++k) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

void initializeMatrix(float *mat, int n) {
    for (int i = 0; i < n * n; i++) {
        mat[i] = static_cast<float>(i % 10);
    }
}

void printMatrix(float *mat, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            std::cout << mat[i * n + j] << "\t";
        }
        std::cout << std::endl;
    }
}

int main() {
    int size = N * N * sizeof(float);
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    initializeMatrix(h_A, N);
    initializeMatrix(h_B, N);

    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + 15) / 16, (N + 15) / 16);
    matMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    std::cout << "Matrix A:\n";
    printMatrix(h_A, N);

    std::cout << "\nMatrix B:\n";
    printMatrix(h_B, N);

    std::cout << "\nMatrix C (Result):\n";
    printMatrix(h_C, N);

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);

    return 0;
}
